#include "hip/hip_runtime.h"

#include "../header/kernel.h"
#include <eigen3/Eigen/Core>

#include <iostream>
#include <stdio.h>


static void HandleError( hipError_t err, const char *file, int line )
{
	// CUDA error handeling from the "CUDA by example" book
	if (err != hipSuccess)
    {
		printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
		exit( EXIT_FAILURE );
	}
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

// CUDA Version
namespace Kernel
{
    __global__ void cu_dot(Eigen::Vector3d *v1, Eigen::Vector3d *v2, double *out, size_t N)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if(idx < N)
        {
            out[idx] = v1[idx].dot(v2[idx]);
        }
        return;
    }

    // The wrapper for the calling of the actual kernel
    double dot(const std::vector<Eigen::Vector3d> & v1, const std::vector<Eigen::Vector3d> & v2)
    {        
        int n = v1.size();
        double *ret = new double[n];
        // Allocate device arrays
        Eigen::Vector3d *dev_v1, *dev_v2;
        HANDLE_ERROR(hipMalloc((void **)&dev_v1, sizeof(Eigen::Vector3d)*n));
        HANDLE_ERROR(hipMalloc((void **)&dev_v2, sizeof(Eigen::Vector3d)*n));
        double* dev_ret;
        HANDLE_ERROR(hipMalloc((void **)&dev_ret, sizeof(double)*n));

        // Copy to device
        HANDLE_ERROR(hipMemcpy(dev_v1, v1.data(), sizeof(Eigen::Vector3d)*n, hipMemcpyHostToDevice));
        HANDLE_ERROR(hipMemcpy(dev_v2, v2.data(), sizeof(Eigen::Vector3d)*n, hipMemcpyHostToDevice));

        // Dot product
        cu_dot<<<(n+1023)/1024, 1024>>>(dev_v1, dev_v2, dev_ret, n);
        
        // Copy to host
        HANDLE_ERROR(hipMemcpy(ret, dev_ret, sizeof(double)*n, hipMemcpyDeviceToHost));

        // Reduction of the array
        for (int i=1; i<n; ++i)
        {
            ret[0] += ret[i];
        }

        // Return
        return ret[0];
    }

    /*-------------------- dot product matrix ---------------------*/
    __global__ void cu_dotMatrix(const Eigen::ArrayXXd *m1, const Eigen::ArrayXXd *m2, Eigen::ArrayXXd *out, size_t N)
    {
        int ROW = blockIdx.y*blockDim.y+threadIdx.y;
        int COL = blockIdx.x*blockDim.x+threadIdx.x;      

        double tmpSum = 0;
        
        if (ROW < N && COL < N) {
            // each thread computes one element of the block sub-matrix
            for (int i = 0; i < N; i++) {
                tmpSum += (*m1)(ROW * N + i) * (*m2)(i * N + COL);
            }
        }
        (*out)(ROW * N + COL) = tmpSum; 
        return;
    }
    
    // The wrapper for the calling of the actual kernel
    Eigen::MatrixXd dotMatrix(const Eigen::ArrayXXd & m1, const Eigen::ArrayXXd  & m2)
    {        
        int n1 = m1.size();
        int n2 = m2.size();

        //Instantiate
        Eigen::ArrayXXd *dev_m1, *dev_m2;
        //Alloc on GPU 
        HANDLE_ERROR(hipMalloc((void **)&dev_m1, sizeof(double)*n1));
        HANDLE_ERROR(hipMalloc((void **)&dev_m2, sizeof(double)*n2));
        
        //Instantiate
        Eigen::ArrayXXd* dev_ret;
        //Alloc on GPU
        HANDLE_ERROR(hipMalloc((void **)&dev_ret, sizeof(double)*n2));

        // Copy to device
        HANDLE_ERROR(hipMemcpy(dev_m1, m1.data(), sizeof(double)*n1, hipMemcpyHostToDevice));
        HANDLE_ERROR(hipMemcpy(dev_m2, m2.data(), sizeof(double)*n2, hipMemcpyHostToDevice));
    
        // Dot product
        cu_dotMatrix<<<(n1+1023)/1024, 1024>>>(dev_m1, dev_m2, dev_ret, n1);
                
        // Copy to host
        //HANDLE_ERROR(hipMemcpy(ret, dev_ret, sizeof(double)*n, hipMemcpyDeviceToHost));


        return m1;
    }
}

